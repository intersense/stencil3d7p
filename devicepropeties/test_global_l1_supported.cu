//test_global_l1_supported.cu
#include <stdio.h>
#include <hip/hip_runtime.h>


// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

int main(){
    hipDeviceProp_t prop;
    checkCuda(hipGetDeviceProperties(&prop,0));
    if (prop.globalL1CacheSupported)
        printf("Global L1 Cache Supported\n");
    else
        printf("Global L1 Cache NOT Supported\n");

    if (prop.localL1CacheSupported)
        printf("Local L1 Cache Supported\n");
    else
        printf("Local L1 Cache NOT Supported\n");
    return 0;
    }