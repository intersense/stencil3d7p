//jacobi7.cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_call1.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include "getopt.h"
#include "include/jacobi7_cuda.h"
#include "include/jacobi7.h"

int main(int argc, char* *argv){
    if(argc != 7) {
        printf("USAGE: %s <NX> <NY> <NZ> <TX> <TY> <TIME STEPS>\n", argv[0]);
        return 1;
    }
    // program parameters trans
    const int nx = atoi(argv[1]);
    const int ny = atoi(argv[2]);
    const int nz = atoi(argv[3]);
    const int tx = atoi(argv[4]);
    const int ty = atoi(argv[5]);
    const int timesteps = atoi(argv[6]);
    
    const int xyz = nx * ny * nz;
    const int xyz_byetes = xyz * sizeof(float);

    float *h_dA;
    float *h_dB;
    float *d_dA;
    float *d_dB;

    float *h_dA1;
    float *h_dB1;
    
    // Allocate host buffers
    h_dA = (float*) malloc(xyz_byetes);
    h_dB = (float*) malloc(xyz_byetes);
    h_dA1 = (float*) malloc(xyz_byetes);
    h_dB1 = (float*) malloc(xyz_byetes);

    // grid data iniatialization   
    // randomly generaed test data
    srand(time(NULL));
    int i = 0;
    for(; i < xyz; i++) {
        h_dA[i] = 1 + (float)rand() / (float)RAND_MAX;
        h_dB[i] =  h_dA[i];
        h_dA1[i] = h_dA[i];
        h_dB1[i] = h_dA[i];
    }
    printf("Start computing...");
    printf("h_dB[%d]:%f\n", 2+32*(3+32*4), h_dB[2+32*(3+32*4)]);
    printf("h_dA[%d]:%f\n", 2+32*(3+32*4), h_dA[2+32*(3+32*4)]);

    float *B = 0;
    const int ldb = 0;
    const int ldc = 0;
    

    // Always use device 0
    hipSetDevice(0);

    /* set the ratio of cache/shared memory
    hipFuncCachePreferNone: Default function cache configuration, no preference
    hipFuncCachePreferShared: Prefer larger shared memory and smaller L1 cache
    hipFuncCachePreferL1: Prefer larger L1 cache and smaller shared memory
    */
    //CHECK_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferShared));

    // Allocate device buffers
    CHECK_CALL(hipMalloc((void**)&d_dA, xyz_byetes));
    CHECK_CALL(hipMalloc((void**)&d_dB, xyz_byetes));
    
    // Copy to device
    CHECK_CALL(hipMemcpy(d_dA, h_dA, xyz_byetes, hipMemcpyHostToDevice));
    //CHECK_CALL(hipMemcpy(d_dB, h_dB, xyz_byetes, hipMemcpyHostToDevice));
    CHECK_CALL(hipMemcpy(d_dB, d_dA, xyz_byetes, hipMemcpyDeviceToDevice));
    
    // Setup the kernel
    float* input = d_dA;
    float* output = d_dB;
    dim3 grid(nx/tx, ny/ty);
    dim3 block(tx, ty);

    // Run the kernel
    
    float *tmp;
    float *tmp1;
    float fac = 6.0/(h_dA[0] * h_dA[0]);

    // Run the GPU kernel
    for(int t = 0; t < timesteps; t += 1) {
        jacobi3d_7p_glmem<<<grid, block>>>(input, output, nx, ny, nz, fac);
        // swap input and output
        tmp = input;
        input =  output;
        output = tmp;
    }
    
    // Copy the result to main memory
    CHECK_CALL(hipMemcpy(h_dB, input, xyz_byetes, hipMemcpyDeviceToHost));
    
    // Free buffers
    free(h_dA);
    free(h_dB);
    free(h_dA1);
    free(h_dB1);
    CHECK_CALL(hipFree(d_dA));
    CHECK_CALL(hipFree(d_dB));

}