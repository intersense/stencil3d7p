#include "hip/hip_runtime.h"
//jacobi7.cu
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <jacobi7_cuda_shared.h>
#include <jacobi7.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

int main(int argc, char* *argv){
    if(argc != 7) {
        printf("USAGE: %s <Store_Cached><NX> <NY> <NZ> <TX> <TY> <TIME STEPS>\n", argv[0]);
        return 1;
    }
    // program parameters trans
    const int nx = atoi(argv[1]);
    const int ny = atoi(argv[2]);
    const int nz = atoi(argv[3]);
    const int tx = atoi(argv[4]);
    const int ty = atoi(argv[5]);
    const int timesteps = atoi(argv[6]);
    
    const int xyz = nx * ny * nz;
    const int xyz_bytes = xyz * sizeof(float);

    float *h_A, *h_A1;
    float *h_B, *h_B1;
    float *d_A;
    float *d_B;
    
    int devId = 0;
    hipDeviceProp_t prop;
    checkCuda( hipGetDeviceProperties(&prop, devId));
    //printf("Device : %s\n", prop.name);
    checkCuda( hipSetDevice(devId));
    
    // Allocate host buffers
    checkCuda(hipHostMalloc((void**)&h_A, xyz_bytes)); // host pinned
    checkCuda(hipHostMalloc((void**)&h_B, xyz_bytes));
    
    // for comparison btw CPU and GPU version
    checkCuda(hipHostMalloc((void**)&h_A1, xyz_bytes));
    checkCuda(hipHostMalloc((void**)&h_B1, xyz_bytes));

    // grid data iniatialization   
    // randomly generaed test data
    srand(time(NULL));
    int i = 0;
    for(; i < xyz; i++) {
        h_A[i] = 1 + (float)rand() / (float)RAND_MAX;
        h_A1[i] = h_B1[i] = h_B[i] =  h_A[i];
    }
    
    // A simple comparison of the result
    /*int testIndex = 3 + 3*nx+ 3*nx*ny;
    printf("Iniatialized data[%d]=%f\n", testIndex , h_A[testIndex]);
    printf("h_A[%d]=%f\n", testIndex, h_A[testIndex]);
    printf("h_B[%d]=%f\n", testIndex, h_B[testIndex]);
    printf("h_A1[%d]=%f\n", testIndex, h_A1[testIndex]);
    printf("h_B1[%d]=%f\n", testIndex, h_B1[testIndex]);
    */
    const float fac = 6.0/(h_A[0] * h_A[0]);
    float *tmp;

    // modify nx/tx and ny/ty to (nx+tx-1)/tx and (ny+ty-1)/ty
    // inorder to avoid wrong configuration
    dim3 grid((nx+tx-1)/tx, (ny+ty-1)/ty);
    dim3 block(tx, ty);

    printf("grid:(%d, %d)\n", grid.x, grid.y);
    printf("block:(%d, %d)\n", tx, ty);
    float ms; // elapsed time in milliseconds
    //printf("Start computing...\n");   

    /* set the ratio of cache/shared memory
    hipFuncCachePreferNone: Default function cache configuration, no preference
    hipFuncCachePreferShared: Prefer larger shared memory and smaller L1 cache
    hipFuncCachePreferL1: Prefer larger L1 cache and smaller shared memory
     
    checkCuda(hipDeviceSetCacheConfig(hipFuncCachePreferShared));
   */
    // set the shared memory bank size to eight bytes
    //checkCuda(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));
    
    // k-1, k, and k+1 planes are stored in shmem
    const int sharedMemSize = 3 * (block.x + 2) * (block.y + 2) * sizeof(float); 
    printf("Shared Memory Size: %dKB\n", sharedMemSize>>10);
    // create events and streams
    hipEvent_t startEvent, stopEvent, startEvent1, stopEvent1;
    
    checkCuda( hipEventCreate(&startEvent));
    checkCuda( hipEventCreate(&stopEvent));
    checkCuda( hipEventCreate(&startEvent1));
    checkCuda( hipEventCreate(&stopEvent1));

    // timing start include data transfer and memory allocation
    checkCuda( hipEventRecord(startEvent,0));
    
    // Allocate device buffers
    checkCuda(hipMalloc((void**)&d_A, xyz_bytes)); // device
    checkCuda(hipMalloc((void**)&d_B, xyz_bytes));

    float* input = d_A;
    float* output = d_B;
    
    // copy data to device
    checkCuda( hipMemcpy(d_A, h_A, xyz_bytes, hipMemcpyHostToDevice));
    checkCuda( hipMemcpy(d_B, d_A, xyz_bytes, hipMemcpyDeviceToDevice));
    
    // timing start pure gpu computing
    checkCuda( hipEventRecord(startEvent1, 0));

    // Run the GPU kernel
    for(int t = 0; t < timesteps; t += 1) {
        jacobi3d_7p_25d<<<grid, block, sharedMemSize>>>(input, output, nx, ny, nz, fac);
        tmp = input;
        input =  output;
        output = tmp;
    }
    // timing end pure gpu computing
    checkCuda( hipEventRecord(stopEvent1, 0));
    checkCuda( hipEventSynchronize(stopEvent1));
    checkCuda( hipEventElapsedTime(&ms1, startEvent1, stopEvent1));

    printf("Time of shared memory version (pure GPU) (ms): %f\n", ms1);
  
    double gflop = (xyz * 1e-9) * 7.0 * timesteps;
    double gflop_per_sec = gflop * 1e3 / ms1;
    printf("(GPU) %lf GFlop/s\n", gflop_per_sec);
    double mupdate_per_sec = ((xyz >> 20) * timesteps) * 1e3 / ms1;
    printf("(GPU) %lf M updates/s\n", mupdate_per_sec);

    checkCuda( hipMemcpy(h_A, input, xyz_bytes, hipMemcpyDeviceToHost));

    checkCuda( hipEventRecord(stopEvent, 0));
    checkCuda( hipEventSynchronize(stopEvent));
    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent));
    float *gpuResult = h_A;

    printf("Time of shared memory version (ms): %f\n", ms);
    printf("(including data transfer and memory allocation in GPU.)\n");
  
    gflop = (xyz * 1e-9) * 7.0 * timesteps;
    gflop_per_sec = gflop * 1e3 / ms;
    printf("(GPU) %lf GFlop/s\n", gflop_per_sec);
    mupdate_per_sec = ((xyz >> 20) * timesteps) * 1e3 / ms;
    printf("(GPU) %lf M updates/s\n", mupdate_per_sec);

    // Run the CPU version
    //float startTime = rtclock();
    float *tmp1;
    for(int t = 0; t < timesteps; t += 1) {
        jacobi7(nx, ny, nz, h_A1, h_B1, fac);
        tmp1 = h_A1;
        h_A1 = h_B1;
        h_B1 = tmp1;
    }
    float *cpuResult;
    cpuResult = h_A1;
    
    /*float endTime = rtclock();
    double elapsedTimeC = endTime - startTime;

    printf("Elapsed Time:%lf\n", elapsedTimeC);
    flops = xyz * 7.0 * timesteps;
    gflops = flops / elapsedTimeC / 1e9;
    printf("(CPU) %lf GFlop/s\n", gflops);
    */

    // compare the results btw CPU and GPU version
    double errorNorm, refNorm, diff;
    errorNorm = 0.0;
    refNorm = 0.0;
    i = 0;
    for (; i < xyz; ++i){
        diff = cpuResult[i] - gpuResult[i];
        errorNorm += diff * diff;
        refNorm += cpuResult[i] * cpuResult[i];
        if (abs(diff)> 1e-4)
        {
            printf("GPU[%d]=%f\n", i, gpuResult[i]);
            printf("CPU[%d]=%f\n", i, cpuResult[i]);
        }

    }
    errorNorm = sqrt(errorNorm);
    refNorm   = sqrt(refNorm);

    printf("Error Norm:%lf\n", errorNorm);
    printf("Ref Norm:%lf\n", refNorm);
  
    if(abs(refNorm) < 1e-7) {
      printf("Correctness, FAILED\n");
    }
    else if((errorNorm / refNorm) > 1e-2) {
      printf("Correctness, FAILED\n");
    }
    else {
      printf("Correctness, PASSED\n");
    }
    /*printf("GPU[%d]=%f\n", testIndex, gpuResult[testIndex]);
    printf("CPU[%d]=%f\n", testIndex, cpuResult[testIndex]);
    printf("h_A[%d]=%f\n", testIndex, h_A[testIndex]);
    printf("h_B[%d]=%f\n", testIndex, h_B[testIndex]);
    printf("h_A1[%d]=%f\n", testIndex, h_A1[testIndex]);
    printf("h_B1[%d]=%f\n", testIndex, h_B1[testIndex]);
    
    testIndex = 2 + 2*nx+ 2*nx*ny;
    printf("GPU[%d]=%f\n", testIndex, gpuResult[testIndex]);
    printf("CPU[%d]=%f\n", testIndex, cpuResult[testIndex]);
    printf("h_A[%d]=%f\n", testIndex, h_A[testIndex]);
    printf("h_B[%d]=%f\n", testIndex, h_B[testIndex]);
    printf("h_A1[%d]=%f\n", testIndex, h_A1[testIndex]);
    printf("h_B1[%d]=%f\n", testIndex, h_B1[testIndex]);
    */
    // cleanup
    checkCuda( hipEventDestroy(startEvent));
    checkCuda( hipEventDestroy(stopEvent));
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_A1);
    hipHostFree(h_B1);
    hipFree(d_A);
    hipFree(d_B);

    return 0;

}