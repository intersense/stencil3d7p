#include "hip/hip_runtime.h"
//jacobi7.cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include "getopt.h"
#include "include/jacobi7_cuda.h"

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

// Timer function
double rtclock(){
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return (tp.tv_sec + tp.tv_usec*1.0e-6);
}

int main(int argc, char* *argv){
    if(argc != 7) {
        printf("USAGE: %s <NX> <NY> <NZ> <TX> <TY> <TIME STEPS>\n", argv[0]);
        return 1;
    }
    // program parameters trans
    const int nx = atoi(argv[1]);
    const int ny = atoi(argv[2]);
    const int nz = atoi(argv[3]);
    const int tx = atoi(argv[4]);
    const int ty = atoi(argv[5]);
    const int timesteps = atoi(argv[6]);
    
    const int xyz = nx * ny * nz;
    const int xyz_bytes = xyz * sizeof(float);

    float *h_dA;
    float *h_dB;
    float *d_dA;
    float *d_dB;
    
    // Allocate host buffers
    checkCuda(hipHostMalloc((void**)&h_dA, xyz_bytes)); // host pinned
    checkCuda(hipHostMalloc((void**)&h_dB, xyz_bytes));

    // grid data iniatialization   
    // randomly generaed test data
    srand(time(NULL));
    int i = 0;
    for(; i < xyz; i++) {
        h_dA[i] = 1 + (float)rand() / (float)RAND_MAX;
        h_dB[i] =  h_dA[i];
    }
    printf("Start computing... \n");   

    // Always use device 0
    checkCuda(hipSetDevice(0));

    /* set the ratio of cache/shared memory
    hipFuncCachePreferNone: Default function cache configuration, no preference
    hipFuncCachePreferShared: Prefer larger shared memory and smaller L1 cache
    hipFuncCachePreferL1: Prefer larger L1 cache and smaller shared memory
    */
    //CHECK_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferShared));

    // Allocate device buffers
    checkCuda(hipMalloc((void**)&d_dA, xyz_bytes));
    checkCuda(hipMalloc((void**)&d_dB, xyz_bytes));
    
    hipEvent_t start, stop;
    checkCuda(hipEventCreate(&start));
    checkCuda(hipEventCreate(&stop));
    float milliseconds = 0;

    checkCuda(hipEventRecord(start));
    
    // Copy to device
    checkCuda(hipMemcpy(d_dA, h_dA, xyz_bytes, hipMemcpyHostToDevice));
    checkCuda(hipEventRecord(stop));
    checkCuda(hipEventSynchronize(stop));
    checkCuda(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Data %dMB transferred H2D time:%f\n ms", xyz_bytes >> 20, milliseconds);
    printf("Bandwidth H2D:%f GB/s\n", (float)(xyz_bytes >> 30)/(milliseconds/1e3));

    checkCuda(hipMemcpy(d_dB, d_dA, xyz_bytes, hipMemcpyDeviceToDevice));
    
    // Setup the kernel
    float* input = d_dA;
    float* output = d_dB;
    dim3 grid(nx/tx, ny/ty);
    dim3 block(tx, ty);

    float *tmp;
    float fac = 6.0/(h_dA[0] * h_dA[0]);

    checkCuda(hipEventRecord(start));
    for(int t = 0; t < timesteps; t += 1) {
        jacobi3d_7p_glmem<<<grid, block>>>(input, output, nx, ny, nz, fac);
        tmp = input;
        input =  output;
        output = tmp;
    }
    checkCuda(hipEventRecord(stop));
    checkCuda(hipEventSynchronize(stop));
    
    checkCuda(hipEventElapsedTime(&milliseconds, start, stop));

    printf("Elapsed Time:%f\n ms", milliseconds);
    double flops = xyz * 7.0 * timesteps;
    double gflops = flops / milliseconds / 1e9;
    printf("(GPU) %lf GFlop/s\n", gflops);
    
    // Copy the result to main memory
    checkCuda(hipEventRecord(start));
    if(timesteps%2==0)
        checkCuda(hipMemcpy(h_dB, output, xyz_bytes, hipMemcpyDeviceToHost));
    else
        checkCuda(hipMemcpy(h_dB, input, xyz_bytes, hipMemcpyDeviceToHost));
    checkCuda(hipEventRecord(stop));
    checkCuda(hipEventSynchronize(stop));
    checkCuda(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Data %dMB transferred D2H time:%f\n ms", xyz_bytes >> 20, milliseconds);
    printf("Bandwidth D2H:%f GB/s\n", (float)(xyz_bytes >> 30)/(milliseconds/1e3));
    

    // Run the CPU version
    /*float startTime = rtclock();
    for(int t = 0; t < timesteps; t += 1) {
        jacobi7(nx, ny, nz, h_dA1, h_dB1, fac);
        tmp1 = h_dA1;
        h_dA1 = h_dB1;
        h_dB1 = tmp1;
    }
    float endTime = rtclock();
    double elapsedTimeC = endTime - startTime;

    printf("Elapsed Time:%lf\n", elapsedTimeC);
    flops = xyz * 7.0 * timesteps;
    gflops = flops / elapsedTimeC / 1e9;
    printf("(CPU) %lf GFlop/s\n", gflops);


    // compare the results btw CPU and GPU version
    double errorNorm, refNorm, diff;
    errorNorm = 0.0;
    refNorm = 0.0;
    for (; i < xyz; ++i){
        diff = h_dA1[i] - h_dB[i];
        errorNorm += diff * diff;
        refNorm += h_dA1[i] * h_dA1[i];
        /*if (h_dB[i+nx*(j+ny*k)] != h_dA1[i+nx*(j+ny*k)])
                   diff = 1;*/
    /*}
    errorNorm = sqrt(errorNorm);
    refNorm   = sqrt(refNorm);

    printf("Error Norm:%lf\n", errorNorm);
    printf("Ref Norm:%lf\n", refNorm);
  
    if(abs(refNorm) < 1e-7) {
      printf("Correctness, FAILED\n");
    }
    else if((errorNorm / refNorm) > 1e-2) {
      printf("Correct  ness, FAILED\n");
    }
    else {
      printf("Correctness, PASSED\n");
    }

    /*printf("h_dB[%d]:%f\n", 2+ny*(3+nz*4), h_dB[2+ny*(3+nz*4)]);
    printf("h_dA[%d]:%f\n", 2+ny*(3+nz*4), h_dA[2+ny*(3+nz*4)]);
    printf("h_dB1[%d]:%f\n", 2+ny*(3+nz*4), h_dB1[2+ny*(3+nz*4)]);
    printf("h_dA1[%d]:%f\n", 2+ny*(3+nz*4), h_dA1[2+ny*(3+nz*4)]);
    printf("-----------------------------------\n");
    printf("h_dB[%d]:%f\n", 3+ny*(4+nz*5), h_dB[3+ny*(4+nz*5)]);
    printf("h_dA[%d]:%f\n", 3+ny*(4+nz*5), h_dA[3+ny*(4+nz*5)]);
    printf("h_dB1[%d]:%f\n", 3+ny*(4+nz*5), h_dB1[3+ny*(4+nz*5)]);
    printf("h_dA1[%d]:%f\n", 3+ny*(4+nz*5), h_dA1[3+ny*(4+nz*5)]);
    */
    
    // Free buffers
    hipHostFree(h_dA);
    hipHostFree(h_dB);
    hipFree(d_dA);
    hipFree(d_dB);
    return 0;
}