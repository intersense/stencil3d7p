#include "hip/hip_runtime.h"
//jacobi7.cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include "getopt.h"
#include "jacobi7_cuda.h"
#include "jacobi7.h"

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

// Timer function
double rtclock(){
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return (tp.tv_sec + tp.tv_usec*1.0e-6);
}

int main(int argc, char* *argv){
    if(argc != 8) {
        printf("USAGE: %s <0row_or_1col_first> <NX> <NY> <NZ> <TX> <TY> <TIME STEPS>\n", argv[0]);
        return 1;
    }
    // program parameters trans
    const int row_or_col = atoi(argv[1]);
    const int nx = atoi(argv[2]);
    const int ny = atoi(argv[3]);
    const int nz = atoi(argv[4]);
    const int tx = atoi(argv[5]);
    const int ty = atoi(argv[6]);
    const int timesteps = atoi(argv[7]);
    
    void (*kernel)(float *, float *, const int , const int , const int , float );
    // the first arg determins the row first or column first
    // 0: row first; 1: column first
    if (row_or_col == 0)
        kernel = &jacobi3d_7p_glmem;
    if (row_or_col == 1)
        kernel = &jacobi3d_7p_glmem_col; 
    if (row_or_col == 2)
        kernel = &jacobi3d_7p_glmem_col_row; 

    const int xyz = nx * ny * nz;
    const int xyz_bytes = xyz * sizeof(float);

    float *h_dA, *h_dA1;
    float *h_dB, *h_dB1;
    float *d_dA;
    float *d_dB;
    
    // Allocate host buffers
    checkCuda(hipHostMalloc((void**)&h_dA, xyz_bytes)); // host pinned
    checkCuda(hipHostMalloc((void**)&h_dB, xyz_bytes));
    // for comparison btw CPU and GPU version
    checkCuda(hipHostMalloc((void**)&h_dA1, xyz_bytes));
    checkCuda(hipHostMalloc((void**)&h_dB1, xyz_bytes));

    // grid data iniatialization   
    // randomly generaed test data
    srand(time(NULL));
    int i = 0;
    for(; i < xyz; i++) {
        h_dA[i] = 1 + (float)rand() / (float)RAND_MAX;
        h_dA1[i] = h_dB1[i] = h_dB[i] =  h_dA[i];
    }
    printf("Start computing... \n");   

    // Always use device 0
    checkCuda(hipSetDevice(0));

    /* set the ratio of cache/shared memory
    hipFuncCachePreferNone: Default function cache configuration, no preference
    hipFuncCachePreferShared: Prefer larger shared memory and smaller L1 cache
    hipFuncCachePreferL1: Prefer larger L1 cache and smaller shared memory
    */
    //checkCuda(hipDeviceSetCacheConfig(hipFuncCachePreferShared));

    // Allocate device buffers
    checkCuda(hipMalloc((void**)&d_dA, xyz_bytes));
    checkCuda(hipMalloc((void**)&d_dB, xyz_bytes));
    
    hipEvent_t start, stop;
    checkCuda(hipEventCreate(&start));
    checkCuda(hipEventCreate(&stop));
    float milliseconds = 0;

    checkCuda(hipEventRecord(start));
    
    // Copy to device
    checkCuda(hipMemcpy(d_dA, h_dA, xyz_bytes, hipMemcpyHostToDevice));
    checkCuda(hipEventRecord(stop));
    checkCuda(hipEventSynchronize(stop));
    checkCuda(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Data %dMB transferred H2D time:%f ms\n", xyz_bytes >> 20, milliseconds);
    printf("Bandwidth H2D:%f MB/s\n", (float)(xyz_bytes >> 20)/(milliseconds/1000));

    checkCuda(hipMemcpy(d_dB, d_dA, xyz_bytes, hipMemcpyDeviceToDevice));
    
    // Setup the kernel
    float* input = d_dA;
    float* output = d_dB;
    // modify nx/tx and ny/ty to (nx+tx-1)/tx and (ny+ty-1)/ty
    // inorder to avoid wrong configuration
    dim3 grid((nx+tx-1)/tx, (ny+ty-1)/ty);
    dim3 block(tx, ty);
    printf("grid:(%d, %d)\n", grid.x, grid.y);
    printf("block:(%d, %d)\n", tx, ty);

    float *tmp;
    float fac = 6.0/(h_dA[0] * h_dA[0]);

    checkCuda(hipEventRecord(start));
    for(int t = 0; t < timesteps; t += 1) {
        kernel<<<grid, block>>>(input, output, nx, ny, nz, fac);
        tmp = input;
        input =  output;
        output = tmp;
    }
    checkCuda(hipEventRecord(stop));
    checkCuda(hipEventSynchronize(stop));
    
    checkCuda(hipEventElapsedTime(&milliseconds, start, stop));

    printf("GPU kernel Elapsed Time (pure GPU):%f ms\n", milliseconds);
    double gflop = (xyz * 1e-9) * 7.0 * timesteps;
    double gflop_per_sec = gflop * 1e3 / milliseconds;
    printf("(GPU) %lf GFlop/s\n", gflop_per_sec);
    double mupdate_per_sec = ((xyz >> 20) * timesteps) * 1e3 / milliseconds;
    printf("(GPU) %lf M updates/s\n", mupdate_per_sec);
    
    float *gpuResult;
    // Copy the result to main memory
    if(timesteps%2==0){
        checkCuda(hipEventRecord(start));
        checkCuda(hipMemcpy(h_dB, output, xyz_bytes, hipMemcpyDeviceToHost));
        checkCuda(hipEventRecord(stop));
        checkCuda(hipEventSynchronize(stop));
        checkCuda(hipEventElapsedTime(&milliseconds, start, stop));
        printf("Data %dMB transferred D2H time:%f ms\n", xyz_bytes >> 20, milliseconds);
        printf("Bandwidth D2H:%f MB/s\n", (float)(xyz_bytes >> 20)/(milliseconds/1000));
        gpuResult =  h_dB;
    }
    else{
        checkCuda(hipEventRecord(start));
        checkCuda(hipMemcpy(h_dA, input, xyz_bytes, hipMemcpyDeviceToHost));
        checkCuda(hipEventRecord(stop));
        checkCuda(hipEventSynchronize(stop));
        checkCuda(hipEventElapsedTime(&milliseconds, start, stop));
        printf("Data %dMB transferred D2H time:%f ms\n", xyz_bytes >> 20, milliseconds);
        printf("Bandwidth D2H:%f MB/s\n", (float)(xyz_bytes >> 20)/(milliseconds/1000));
        gpuResult = h_dA;
    }
    
    

    // Run the CPU version
    //float startTime = rtclock();
    float *tmp1;
    for(int t = 0; t < timesteps; t += 1) {
        jacobi7(nx, ny, nz, h_dA1, h_dB1, fac);
        tmp1 = h_dA1;
        h_dA1 = h_dB1;
        h_dB1 = tmp1;
    }
    float *cpuResult;
    if (timesteps%2 == 0)
        cpuResult = h_dB1;
    else
        cpuResult = h_dA1;
    /*float endTime = rtclock();
    double elapsedTimeC = endTime - startTime;

    printf("Elapsed Time:%lf\n", elapsedTimeC);
    flops = xyz * 7.0 * timesteps;
    gflops = flops / elapsedTimeC / 1e9;
    printf("(CPU) %lf GFlop/s\n", gflops);
    */

    // compare the results btw CPU and GPU version
    double errorNorm, refNorm, diff;
    errorNorm = 0.0;
    refNorm = 0.0;
    i = 0;
    for (; i < xyz; ++i){
        diff = cpuResult[i] - gpuResult[i];
        errorNorm += diff * diff;
        refNorm += cpuResult[i] * cpuResult[i];
        if (gpuResult[i] != gpuResult[i])
                   diff = 1;
    }
    errorNorm = sqrt(errorNorm);
    refNorm   = sqrt(refNorm);

    printf("Error Norm:%lf\n", errorNorm);
    printf("Ref Norm:%lf\n", refNorm);
  
    if(abs(refNorm) < 1e-7) {
      printf("Correctness, FAILED\n");
    }
    else if((errorNorm / refNorm) > 1e-2) {
      printf("Correct  ness, FAILED\n");
    }
    else {
      printf("Correctness, PASSED\n");
    }

    // Free buffers
    hipHostFree(h_dA);
    hipHostFree(h_dB);
    hipHostFree(h_dA1);
    hipHostFree(h_dB1);
    hipFree(d_dA);
    hipFree(d_dB);
    return 0;
}